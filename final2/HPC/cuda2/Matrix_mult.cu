#include "hip/hip_runtime.h"
#include<stdio.h>
#define SIZE 10

__global__ void MatrixMul(int a[], int b[], int c[], int n) 
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int result = 0;

    	for(int i = 0; i < SIZE ; i++) 
    	{
        	int p = *(a + ty*SIZE + i);
        	int q = *(b + i*SIZE + tx);
        	result = result + (p*q);
    	}

    	*(c + ty*SIZE + tx) = result;
}
int main() 
{    
	time_t t;
	srand((unsigned) time(&t));
	
	int *a, *b, *c;
	
	a = (int*)malloc(SIZE * SIZE * sizeof(int));
	b = (int*)malloc(SIZE * SIZE * sizeof(int));
	c = (int*)malloc(SIZE * SIZE * sizeof(int));
	
    	for(int i = 0; i < SIZE ; i++) 
    	{
        	for(int j = 0; j < SIZE ; j++) 
        	{
        		*(a + i*SIZE + j) = i;
        		*(b + i*SIZE + j) = i+1;
        	}
    	}
    
	int *d_a, *d_b, *d_c;

    	hipMalloc(&d_a, SIZE * SIZE * sizeof(int));
    	hipMalloc(&d_b, SIZE * SIZE * sizeof(int));
    	hipMalloc(&d_c, SIZE * SIZE * sizeof(int));
    	
    	hipMemcpy(d_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(d_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    	MatrixMul <<<1, SIZE*SIZE>>> (d_a, d_b, d_c, SIZE);
    	
    	hipDeviceSynchronize();

    	hipMemcpy(c, d_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);
    
    	printf("1st matrix: \n");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(a + i*SIZE + j));
		}
		printf("\n");
	}
	printf("2nd matrix: \n");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(b + i*SIZE + j));
		}
		printf("\n");
	}
	printf("Product: \n");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(c + i*SIZE + j));
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(a);
	free(b);
	free(c);

   
    	return 0;
}
