#include <stdio.h>
# include "hip/hip_runtime.h"

#define SIZE 50

__global__ void VectorAdd(int a[], int b[], int c[], int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < n)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int *a, *b, *c;
	
	a = (int*)malloc(SIZE * sizeof(int));
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));
	
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i+1;
		b[i] = i;
	}
	
	int *d_a, *d_b, *d_c;
	
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
		
	VectorAdd <<< 2, SIZE/2 >>> (d_a, d_b, d_c, SIZE);
	
	hipDeviceSynchronize(); // jab tak saare threads ka kaam nahi hota.... tab tak ruko
	
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(a);
	free(b);
	free(c);

	return 0;
}
