#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 128

//Note: N should always be in powers of 2 (like 2, 4, 8, 16, 32, ...) -Mohit Agrawal

__global__ void FindAvg(int* input)
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			input[fst] += input[snd];
		}
		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}
int main()
{
	time_t t;
	srand((unsigned) time(&t));
	
	int *h;
	h = (int*)malloc(N*sizeof(int));
	
	for(int i=0; i<N; i++)
	{
		h[i] = rand()%N;
	}
	for(int i=0; i<N; i++)
	{
		printf("%d ", h[i]);
	}
	printf("\n");

	int* d;
	hipMalloc(&d, N*sizeof(int));
	
	hipMemcpy(d, h, N*sizeof(int), hipMemcpyHostToDevice);

	FindAvg <<<1, N/2 >>>(d);
	
	hipDeviceSynchronize();

	int *result;
	result = (int*)malloc(sizeof(int));
	
	hipMemcpy(result, d, sizeof(int), hipMemcpyDeviceToHost);

	float avg = (float)result[0]/N;
	printf("Average is: %f \n", avg);

	hipFree(d);
	free(h);

	return 0;
}
